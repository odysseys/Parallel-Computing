#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include <stdio.h>
#define TILE_WIDTH 2
#define BLOCK_SIZE 32


namespace cuda
{
  __global__
  void
  matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    //For repeatly access blocks, use shared memory to speed up
    __shared__ float local_mat_1[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float local_mat_2[BLOCK_SIZE][BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int block_offsetx = blockIdx.x * BLOCK_SIZE;
    int block_offsety = blockIdx.y * BLOCK_SIZE;
    float sum = 0.0f;
    
    #pragma unroll
    for(int i = 0; i < sq_dimension; i += BLOCK_SIZE)
    {
      //Transfer to 2-D matrix to avoid memory bank conflict
      //local_mat_1 is row-based moving for each 32*32 block in 1st input matrix
      if(ty + block_offsety < sq_dimension && tx + i < sq_dimension)
        local_mat_1[ty][tx] = sq_matrix_1[(ty + block_offsety) * sq_dimension + tx + i            ];
      else
        local_mat_1[ty][tx] = 0;
      
      //local_mat_2 is column-based moving for each 32*32 block in 2nd input matrix
      if(tx + block_offsetx < sq_dimension && ty + i < sq_dimension)
        local_mat_2[ty][tx] = sq_matrix_2[(ty + i            ) * sq_dimension + tx + block_offsetx];
      else
        local_mat_2[ty][tx] = 0;
      
      //must wait all threads finishing moving data into shared memory
      __syncthreads();
      
      #pragma unroll
      for(int k = 0; k < BLOCK_SIZE; k++)
      {
        sum += local_mat_1[ty][k] * local_mat_2[k][tx];
      }
      
      __syncthreads();//must wait all threads sum up
    }

    if(tx + block_offsetx < sq_dimension && ty + block_offsety < sq_dimension)
    {
      sq_matrix_result[(ty + block_offsety) * sq_dimension + tx + block_offsetx] = sum;
      //calculate the correct position of the product
    }
  }

  void
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;

    /***************************************************
  1st Part: Allocation of memory on device memory
    ****************************************************/

    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);

    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);

    /***************************************************
   2nd Part: Inovke kernel
    ****************************************************/
    //fix the size to maximum number 1024
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //if sq_dimension*sq_dimension>1024, use other blocks to calculate
    int gridx = int (sq_dimension + BLOCK_SIZE - 1) / int(BLOCK_SIZE);
    dim3 dimGrid(gridx, gridx);
    matrix_mul_kernel<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    //, dimBlock.x * dimBlock.x * sizeof(float)
    /***************************************************
   3rd Part: Transfer result from device to host
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }
} // namespace cuda
