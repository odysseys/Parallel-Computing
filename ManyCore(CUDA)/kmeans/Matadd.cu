#include "hip/hip_runtime.h"
#include<cutil_inline.h>  
#include<iostream>  

using namespace std; 

#define N 32

// Kernel definition
__global__ void MatAdd(float A[N], float B[N], float* C)  
{  
    int i = blockIdx.x * blockDim.x + threadIdx.x; //get thread index by built-in variables  
    if (i < N)  
        C[i] = A[i] + B[i];  
}       

int main()  
{  
    float A[N],B[N]; // host variable  
    float *dA, *dB; // device variable, to have same value with A,B  
    float *device_res, *host_res; // device and host result, to be device and host variable respectively  
  
    // initialize host variable  
    memset(A,0,sizeof(A));  
    memset(B,0,sizeof(B));  
    A[0] = 1;  
    B[0] = 2;  
  
  
    // allocate for device variable and set value to them  
    hipMalloc((void**) &dA,N*sizeof(float));  
    hipMalloc((void**) &dB,N*sizeof(float));  
    hipMemcpy(dA, A, N*sizeof(float),hipMemcpyHostToDevice);  
    hipMemcpy(dB, B, N*sizeof(float),hipMemcpyHostToDevice);  
  
    //malloc for host and device variable  
    host_res = (float*) malloc(N*sizeof(float));  
    hipMalloc((void**)&device_res, N*sizeof(float));  
  
    // Kernel invocation  
    int threadsPerBlock = 16;  
    int numBlocks = N/threadsPerBlock;   
    MatAdd<<<numBlocks, threadsPerBlock>>>(dA, dB, device_res);  
  
    hipMemcpy(host_res, device_res, N*sizeof(float),hipMemcpyDeviceToHost); //copy from device to host  
      
    // validate  
    int i;  
    float sum = 0;  
    for(i=0;i<N;i++)  
        sum += host_res[i];  
    cout<<sum<<endl;  
  
    //free variables  
    hipFree(dA);  
    hipFree(dB);  
  
    hipFree(device_res);  
    free(host_res);  
} 